#include "hip/hip_runtime.h"
/* For IDE: */
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <thread>
#include <chrono>

#ifdef _WIN64
#include <windows.h>
#endif

#include "argon2-gpu-common/argon2-common.h"
#include "argon2-cuda/kernels.h"

#include "argon2-cuda/cudaexception.h"

#include <stdexcept>
#ifndef NDEBUG
#include <iostream>

#endif

#define ARGON2_D  0
#define ARGON2_I  1
#define ARGON2_ID 2

#define ARGON2_VERSION_10 0x10
#define ARGON2_VERSION_13 0x13

#define ARGON2_BLOCK_SIZE 1024
#define ARGON2_DWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 4)
#define ARGON2_QWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 8)
#define ARGON2_SYNC_POINTS 4

#define THREADS_PER_LANE 32
#define QWORDS_PER_THREAD (ARGON2_QWORDS_IN_BLOCK / 32)

#define ARGON2_PREHASH_DIGEST_LENGTH	16
#define ARGON2_PREHASH_SEED_LENGTH		18

namespace argon2 {
namespace cuda {

#include "blake2b.cu"

__device__ uint64_t u64_build(uint32_t hi, uint32_t lo)
{
    return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint32_t u64_lo(uint64_t x)
{
    return (uint32_t)x;
}

__device__ uint32_t u64_hi(uint64_t x)
{
    return (uint32_t)(x >> 32);
}

struct u64_shuffle_buf {
    uint32_t lo[THREADS_PER_LANE];
    uint32_t hi[THREADS_PER_LANE];
};

__device__ uint64_t u64_shuffle(uint64_t v, uint32_t thread_src,
                                uint32_t thread, struct u64_shuffle_buf *buf)
{
    uint32_t lo = u64_lo(v);
    uint32_t hi = u64_hi(v);

    buf->lo[thread] = lo;
    buf->hi[thread] = hi;

    __syncthreads();

    lo = buf->lo[thread_src];
    hi = buf->hi[thread_src];

    return u64_build(hi, lo);
}

struct block_g {
    uint64_t data[ARGON2_QWORDS_IN_BLOCK];
};

struct block_th {
    uint64_t a, b, c, d;
};

__device__ uint64_t cmpeq_mask(uint32_t test, uint32_t ref)
{
    uint32_t x = -(uint32_t)(test == ref);
    return u64_build(x, x);
}

__device__ uint64_t block_th_get(const struct block_th *b, uint32_t idx)
{
    uint64_t res = 0;
    res ^= cmpeq_mask(idx, 0) & b->a;
    res ^= cmpeq_mask(idx, 1) & b->b;
    res ^= cmpeq_mask(idx, 2) & b->c;
    res ^= cmpeq_mask(idx, 3) & b->d;
    return res;
}

__device__ void block_th_set(struct block_th *b, uint32_t idx, uint64_t v)
{
    b->a ^= cmpeq_mask(idx, 0) & (v ^ b->a);
    b->b ^= cmpeq_mask(idx, 1) & (v ^ b->b);
    b->c ^= cmpeq_mask(idx, 2) & (v ^ b->c);
    b->d ^= cmpeq_mask(idx, 3) & (v ^ b->d);
}

__device__ void move_block(struct block_th *dst, const struct block_th *src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th *dst, const struct block_th *src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block(struct block_th *dst, const struct block_g *src,
                           uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_xor(struct block_th *dst, const struct block_g *src,
                               uint32_t thread)
{
    dst->a ^= src->data[0 * THREADS_PER_LANE + thread];
    dst->b ^= src->data[1 * THREADS_PER_LANE + thread];
    dst->c ^= src->data[2 * THREADS_PER_LANE + thread];
    dst->d ^= src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void store_block(struct block_g *dst, const struct block_th *src,
                            uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ uint64_t f(uint64_t x, uint64_t y)
{
    uint32_t xlo = u64_lo(x);
    uint32_t ylo = u64_lo(y);
    return x + y + 2 * u64_build(__umulhi(xlo, ylo), xlo * ylo);
}

__device__ void g(struct block_th *block)
{

	asm ("{"
	     ".reg .u64 d1, d2, a, b, c, d;\n\t"
		 ".reg .u32 s1, s2, s3, s4;\n\t"
		 "add.u64 d1, %0, %1;\n\t"
		 "cvt.u32.u64 s1, %0;\n\t"
		 "cvt.u32.u64 s2, %1;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 a, {s3, s4};\n\t"
		 "shl.b64 d2, a, 1;\n\t"
		 "add.u64 a, d1, d2;\n\t"
		 "xor.b64 d1, %3, a;\n\t"
		 "mov.b64 {s1, s2}, d1;\n\t"
		 "mov.b64 d, {s2, s1};\n\t"
		 "add.u64 d1, %2, d;\n\t"
		 "cvt.u32.u64 s1, %2;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 c, {s3, s4};\n\t"
		 "shl.b64 d2, c, 1;\n\t"
		 "add.u64 c, d1, d2;\n\t"
		 "xor.b64 d1, %1, c;\n\t"
		 "mov.b64 {s3, s4}, d1;\n\t"
		 "prmt.b32 s2, s3, s4, 0x6543;\n\t"
		 "prmt.b32 s1, s3, s4, 0x2107;\n\t"
		 "mov.b64 b, {s2, s1};\n\t"
		 "add.u64 d1, a, b;\n\t"
		 "cvt.u32.u64 s1, a;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 a, {s3, s4};\n\t"
		 "shl.b64 d2, a, 1;\n\t"
		 "add.u64 %0, d1, d2;\n\t"
		 "xor.b64 d1, d, %0;\n\t"
		 "mov.b64 {s3, s4}, d1;\n\t"
		 "prmt.b32 s2, s3, s4, 0x5432;\n\t"
		 "prmt.b32 s1, s3, s4, 0x1076;\n\t"
		 "mov.b64 %3, {s2, s1};\n\t"
		 "add.u64 d1, c, %3;\n\t"
		 "cvt.u32.u64 s1, c;\n\t"
		 "mul.lo.u32 s3, s1, s2;\n\t"
		 "mul.hi.u32 s4, s1, s2;\n\t"
		 "mov.b64 c, {s3, s4};\n\t"
		 "shl.b64 d2, c, 1;\n\t"
		 "add.u64 %2, d1, d2;\n\t"
		 "xor.b64 d1, b, %2;\n\t"
		 "shl.b64 a, d1, 1;\n\t"
		 "shr.b64 b, d1, 63;\n\t"
		 "add.u64 %1, a, b;\n\t"
	"}" : "+l"(block->a), "+l"(block->b), "+l"(block->c), "+l"(block->d));

/*
    uint64_t a, b, c, d;
    a = block->a;
    b = block->b;
    c = block->c;
    d = block->d;

 	a = f(a, b);
    d = rotr64(d ^ a, 32);
    c = f(c, d);
    b = rotr64(b ^ c, 24);
    a = f(a, b);
    d = rotr64(d ^ a, 16);
    c = f(c, d);
    b = rotr64(b ^ c, 63);

    block->a = a;
    block->b = b;
    block->c = c;
    block->d = d; */
}

template<class shuffle>
__device__ void apply_shuffle(struct block_th *block, uint32_t thread,
                              struct u64_shuffle_buf *buf)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t src_thr = shuffle::apply(thread, i);

        uint64_t v = block_th_get(block, i);
        v = u64_shuffle(v, src_thr, thread, buf);
        block_th_set(block, i, v);
    }
}

__device__ void transpose(struct block_th *block, uint32_t thread,
                          struct u64_shuffle_buf *buf)
{
    uint32_t thread_group = (thread & 0x0C) >> 2;
    for (uint32_t i = 1; i < QWORDS_PER_THREAD; i++) {
        uint32_t thr = (i << 2) ^ thread;
        uint32_t idx = thread_group ^ i;

        uint64_t v = block_th_get(block, idx);
        v = u64_shuffle(v, thr, thread, buf);
        block_th_set(block, idx, v);
    }
}

struct identity_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return thread;
    }
};

struct shift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct unshift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct shift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

struct unshift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

__device__ void shuffle_block(struct block_th *block, uint32_t thread,
                              struct u64_shuffle_buf *buf)
{
    transpose(block, thread, buf);

    g(block);

    apply_shuffle<shift1_shuffle>(block, thread, buf);

    g(block);

    apply_shuffle<unshift1_shuffle>(block, thread, buf);
    transpose(block, thread, buf);

    g(block);

    apply_shuffle<shift2_shuffle>(block, thread, buf);

    g(block);

    apply_shuffle<unshift2_shuffle>(block, thread, buf);
}

__device__ void next_addresses(struct block_th *addr, struct block_th *tmp,
                               uint32_t thread_input, uint32_t thread,
                               struct u64_shuffle_buf *buf)
{
    addr->a = u64_build(0, thread_input);
    addr->b = 0;
    addr->c = 0;
    addr->d = 0;

    shuffle_block(addr, thread, buf);

    addr->a ^= u64_build(0, thread_input);
    move_block(tmp, addr);

    shuffle_block(addr, thread, buf);

    xor_block(addr, tmp);
}

__device__ void compute_ref_pos(
        uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t lane, uint32_t slice, uint32_t offset,
        uint32_t *ref_lane, uint32_t *ref_index)
{
    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    *ref_lane = *ref_lane % lanes;

    uint32_t base;
    if (pass != 0) {
        base = lane_blocks - segment_blocks;
    } else {
        if (slice == 0) {
            *ref_lane = lane;
        }
        base = slice * segment_blocks;
    }

    uint32_t ref_area_size = base + offset - 1;
    if (*ref_lane != lane) {
        ref_area_size = min(ref_area_size, base);
    }

    *ref_index = __umulhi(*ref_index, *ref_index);
    *ref_index = ref_area_size - 1 - __umulhi(ref_area_size, *ref_index);

    if (pass != 0 && slice != ARGON2_SYNC_POINTS - 1) {
        *ref_index += (slice + 1) * segment_blocks;
        if (*ref_index >= lane_blocks) {
            *ref_index -= lane_blocks;
        }
    }
}

struct ref {
    uint32_t ref_lane;
    uint32_t ref_index;
};

/*
 * Refs hierarchy:
 * lanes -> passes -> slices -> blocks
 */
template<uint32_t type>
__global__ void argon2_precompute_kernel(
        struct ref *refs, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    uint32_t block_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t warp = threadIdx.y;
    uint32_t thread = threadIdx.x;

    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf = &shuffle_bufs[warp];

    uint32_t segment_addr_blocks = (segment_blocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t block = block_id % segment_addr_blocks;
    uint32_t segment = block_id / segment_addr_blocks;

    uint32_t slice, pass, pass_id, lane;
    if (type == ARGON2_ID) {
        slice = segment % (ARGON2_SYNC_POINTS / 2);
        lane = segment / (ARGON2_SYNC_POINTS / 2);
        pass_id = pass = 0;
    } else {
        slice = segment % ARGON2_SYNC_POINTS;
        pass_id = segment / ARGON2_SYNC_POINTS;

        pass = pass_id % passes;
        lane = pass_id / passes;
    }

    struct block_th addr, tmp;

    uint32_t thread_input;
    switch (thread) {
    case 0:
        thread_input = pass;
        break;
    case 1:
        thread_input = lane;
        break;
    case 2:
        thread_input = slice;
        break;
    case 3:
        thread_input = lanes * segment_blocks * ARGON2_SYNC_POINTS;
        break;
    case 4:
        thread_input = passes;
        break;
    case 5:
        thread_input = type;
        break;
    case 6:
        thread_input = block + 1;
        break;
    default:
        thread_input = 0;
        break;
    }

    next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);

    refs += segment * segment_blocks;

    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos = i * THREADS_PER_LANE + thread;
        uint32_t offset = block * ARGON2_QWORDS_IN_BLOCK + pos;
        if (offset < segment_blocks) {
            uint64_t v = block_th_get(&addr, i);
            uint32_t ref_index = u64_lo(v);
            uint32_t ref_lane  = u64_hi(v);

            compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                            &ref_lane, &ref_index);

            refs[offset].ref_index = ref_index;
            refs[offset].ref_lane  = ref_lane;
        }
    }
}

template<uint32_t version>
__device__ void argon2_core(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp,
        struct u64_shuffle_buf *shuffle_buf, uint32_t lanes,
        uint32_t thread, uint32_t pass, uint32_t ref_index, uint32_t ref_lane)
{
    struct block_g *mem_ref = memory + ref_index * lanes + ref_lane;

    if (version != ARGON2_VERSION_10 && pass != 0) {
        load_block(tmp, mem_curr, thread);
        load_block_xor(prev, mem_ref, thread);
        xor_block(tmp, prev);
    } else {
        load_block_xor(prev, mem_ref, thread);
        move_block(tmp, prev);
    }

    shuffle_block(prev, thread, shuffle_buf);

    xor_block(prev, tmp);

    store_block(mem_curr, prev, thread);
}

template<uint32_t type, uint32_t version>
__device__ void argon2_step_precompute(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp,
        struct u64_shuffle_buf *shuffle_buf, const struct ref **refs,
        uint32_t lanes, uint32_t segment_blocks, uint32_t thread,
        uint32_t lane, uint32_t pass, uint32_t slice, uint32_t offset)
{
    uint32_t ref_index, ref_lane;
    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
            slice < ARGON2_SYNC_POINTS / 2)) {
        ref_index = (*refs)->ref_index;
        ref_lane = (*refs)->ref_lane;
        (*refs)++;
    } else {
        uint64_t v = u64_shuffle(prev->a, 0, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);

        compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                        &ref_lane, &ref_index);
    }

    argon2_core<version>(memory, mem_curr, prev, tmp, shuffle_buf, lanes,
                         thread, pass, ref_index, ref_lane);
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment_precompute(
        struct block_g *memory, const struct ref *refs,
        uint32_t passes, uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t slice)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[blockDim.y * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g *mem_segment =
            memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        if (pass == 0 && slice < ARGON2_SYNC_POINTS / 2) {
            refs += lane * (lane_blocks / 2) + slice * segment_blocks;
            refs += start_offset;
        }
    } else {
        refs += (lane * passes + pass) * lane_blocks + slice * segment_blocks;
        refs += start_offset;
    }

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step_precompute<type, version>(
                    memory, mem_curr, &prev, &tmp, shuffle_buf, &refs, lanes,
                    segment_blocks, thread, lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot_precompute(
        struct block_g *memory, const struct ref *refs, uint32_t passes,
        uint32_t lanes, uint32_t segment_blocks)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[lanes * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g *mem_lane = memory + lane;
    struct block_g *mem_prev = mem_lane + 1 * lanes;
    struct block_g *mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        refs += lane * (lane_blocks / 2) + 2;
    } else {
        refs += lane * passes * lane_blocks + 2;
    }

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step_precompute<type, version>(
                            memory, mem_curr, &prev, &tmp, shuffle_buf, &refs,
                            lanes, segment_blocks, thread,
                            lane, pass, slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();
        }

        mem_curr = mem_lane;
    }
}

template<uint32_t type, uint32_t version>
__device__ void argon2_step(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_th *prev, struct block_th *tmp, struct block_th *addr,
        struct u64_shuffle_buf *shuffle_buf, uint32_t lanes,
        uint32_t segment_blocks, uint32_t thread, uint32_t *thread_input,
        uint32_t lane, uint32_t pass, uint32_t slice, uint32_t offset)
{
    uint32_t ref_index, ref_lane;

    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
            slice < ARGON2_SYNC_POINTS / 2)) {
        uint32_t addr_index = offset % ARGON2_QWORDS_IN_BLOCK;
        if (addr_index == 0) {
            if (thread == 6) {
                ++*thread_input;
            }
            next_addresses(addr, tmp, *thread_input, thread, shuffle_buf);
        }

        uint32_t thr = addr_index % THREADS_PER_LANE;
        uint32_t idx = addr_index / THREADS_PER_LANE;

        uint64_t v = block_th_get(addr, idx);
        v = u64_shuffle(v, thr, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);
    } else {
        uint64_t v = u64_shuffle(prev->a, 0, thread, shuffle_buf);
        ref_index = u64_lo(v);
        ref_lane  = u64_hi(v);
    }

    compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                    &ref_lane, &ref_index);

    argon2_core<version>(memory, mem_curr, prev, tmp, shuffle_buf, lanes,
                         thread, pass, ref_index, ref_lane);
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks, uint32_t pass, uint32_t slice)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[blockDim.y * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 0:
            thread_input = pass;
            break;
        case 1:
            thread_input = lane;
            break;
        case 2:
            thread_input = slice;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (pass == 0 && slice == 0 && segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);
        }
    }

    struct block_g *mem_segment =
            memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step<type, version>(
                    memory, mem_curr, &prev, &tmp, &addr, shuffle_buf,
                    lanes, segment_blocks, thread, &thread_input,
                    lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    extern __shared__ struct u64_shuffle_buf shuffle_bufs[];
    struct u64_shuffle_buf *shuffle_buf =
            &shuffle_bufs[lanes * threadIdx.z + threadIdx.y];

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 1:
            thread_input = lane;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread, shuffle_buf);
        }
    }

    struct block_g *mem_lane = memory + lane;
    struct block_g *mem_prev = mem_lane + 1 * lanes;
    struct block_g *mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step<type, version>(
                            memory, mem_curr, &prev, &tmp, &addr, shuffle_buf,
                            lanes, segment_blocks, thread, &thread_input,
                            lane, pass, slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();

            if (type == ARGON2_I || type == ARGON2_ID) {
                if (thread == 2) {
                    ++thread_input;
                }
                if (thread == 6) {
                    thread_input = 0;
                }
            }
        }
        if (type == ARGON2_I) {
            if (thread == 0) {
                ++thread_input;
            }
            if (thread == 2) {
                thread_input = 0;
            }
        }
        mem_curr = mem_lane;
    }
}

__device__ __forceinline__ void argon2_genseed_generic(uint32_t *initHash, uint32_t *seed, int job_id, int thr_id) {
	uint32_t *seed_local = seed + job_id * ARGON2_PREHASH_DIGEST_LENGTH;

	for (int i = 0; i < ARGON2_PREHASH_DIGEST_LENGTH / 4; i++) {
		initHash[i * 4 + thr_id] = seed_local[i * 4 + thr_id];
	}
}

__device__ __forceinline__ void argon2_genseed_crds_dyn_arg(uint32_t *initHash, uint32_t *seed,
		int lanes, int m_cost, int t_cost, int version, int job_id, int thr_id) {
	uint64_t *h = (uint64_t*)&initHash[20];
	uint32_t *buf = (uint32_t*)&h[10];
	uint32_t *value = &buf[32];

	for (int i = 0; i < 5; i++) {
		initHash[i * 4 + thr_id] = seed[i * 4 + thr_id];
	}

	if (thr_id == 3) {
		uint32_t x = seed[19] + job_id;
		uint8_t *p = (uint8_t *) &initHash[19];
		p[3] = x & 0xff;
		p[2] = (x >> 8) & 0xff;
		p[1] = (x >> 16) & 0xff;
		p[0] = (x >> 24) & 0xff;
	}

	int buf_len = blake2b_init(h, ARGON2_PREHASH_DIGEST_LENGTH, thr_id);
	*value = lanes; //lanes
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = 32; //outlen
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = m_cost; //m_cost
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = t_cost; //t_cost
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = version; //version
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = ARGON2_D; //type
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	*value = 80; //pw_len
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	buf_len = blake2b_update(initHash, 20, h, buf, buf_len, thr_id);
	*value = 80; //salt_len
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	buf_len = blake2b_update(initHash, 20, h, buf, buf_len, thr_id);
	*value = 0; //secret_len
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);
	*value = 0; //ad_len
	buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
	buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);

	blake2b_final(initHash, ARGON2_PREHASH_DIGEST_LENGTH, h, buf, buf_len, thr_id);
}

__device__ __forceinline__ void argon2_genseed_urx(uint32_t *initHash, uint32_t *seed, uint32_t *secret, uint32_t secretLen, uint32_t *ad, uint32_t adLen,
                                                            int lanes, int m_cost, int t_cost, int version, int job_id, int thr_id) {
    uint64_t *h = (uint64_t*)&initHash[20];
    uint32_t *buf = (uint32_t*)&h[10];
    uint32_t *value = &buf[32];

    for (int i = 0; i < 5; i++) {
        initHash[i * 4 + thr_id] = seed[i * 4 + thr_id];
    }

    if (thr_id == 3) {
        uint32_t x = seed[19] + job_id;
        uint8_t *p = (uint8_t *) &initHash[19];
        p[3] = x & 0xff;
        p[2] = (x >> 8) & 0xff;
        p[1] = (x >> 16) & 0xff;
        p[0] = (x >> 24) & 0xff;
    }

    int buf_len = blake2b_init(h, ARGON2_PREHASH_DIGEST_LENGTH, thr_id);
    *value = lanes; //lanes
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 32; //outlen
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = m_cost; //m_cost
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = t_cost; //t_cost
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = version; //version
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = ARGON2_D; //type
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 40; //pw_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(initHash, 10, h, buf, buf_len, thr_id);
    *value = 40; //salt_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(&initHash[10], 10, h, buf, buf_len, thr_id);
    *value = secretLen; //secret_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(secret, secretLen / 4, h, buf, buf_len, thr_id);
    *value = adLen; //ad_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(ad, adLen / 4, h, buf, buf_len, thr_id);

    blake2b_final(initHash, ARGON2_PREHASH_DIGEST_LENGTH, h, buf, buf_len, thr_id);
}

__global__ void argon2_kernel_preseed(
		uint32_t algo, struct block_g *memory, uint32_t *seed, uint32_t lanes, uint32_t segment_blocks,
        uint32_t *secret, uint32_t secretLen, uint32_t *ad, uint32_t adLen) {
    int job_id = blockIdx.x;
	int lane_thr = threadIdx.x / 4;
	int thr_id = threadIdx.x % 4;
	int lane = lane_thr % lanes;
	int idx = lane_thr / lanes;

    extern __shared__ uint32_t shared[];

    uint32_t *initHash = &shared[lane_thr * 88];

    if(algo == 1) // Crds
		argon2_genseed_crds_dyn_arg(initHash, seed, lanes, 250, 1, ARGON2_VERSION_10, job_id, thr_id);
    else if(algo == 2) // Dyn
		argon2_genseed_crds_dyn_arg(initHash, seed, lanes, 500, 2, ARGON2_VERSION_10, job_id, thr_id);
    else if(algo == 3) // Arg
		argon2_genseed_crds_dyn_arg(initHash, seed, lanes, 4096, 1, ARGON2_VERSION_13, job_id, thr_id);
    else if(algo == 4) //Urx
        argon2_genseed_urx(initHash, seed, secret, secretLen, ad, adLen, lanes, 512, 1, ARGON2_VERSION_13, job_id, thr_id);
	else
		argon2_genseed_generic(initHash, seed, job_id, thr_id);

	if (thr_id == 0) {
		initHash[ARGON2_PREHASH_DIGEST_LENGTH] = idx;
		initHash[ARGON2_PREHASH_DIGEST_LENGTH + 1] = lane;
	}

    /* select job's memory region: */
    memory += job_id * lanes * ARGON2_SYNC_POINTS * segment_blocks;

	blake2b_digestLong((uint32_t*)(memory + lane + idx * lanes)->data, ARGON2_DWORDS_IN_BLOCK, initHash, ARGON2_PREHASH_SEED_LENGTH, thr_id, &initHash[20]);
}

__global__ void argon2_kernel_finalize(
		struct block_g *memory, uint32_t *out, uint32_t outLen, uint32_t lanes, uint32_t segment_blocks) {
    extern __shared__ uint32_t shared[];

	int job_id = blockIdx.x;
	int thread = threadIdx.x;

	int lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;
	/* select job's memory region: */
	memory += ((job_id + 1) * lanes * lane_blocks - lanes);
	out += job_id * outLen;
	struct block_g *dst = memory;

	for(int l=1;l<lanes;l++) {
		memory += 1;
		for (int i = 0; i < 4/*ARGON2_QWORDS_IN_BLOCK*/; i++) {
			dst->data[thread * 4 + i] ^= memory->data[thread * 4 + i];
		}
	}

	if(thread / 4 == 0) {
		blake2b_digestLong(out, outLen, (uint32_t *) dst, ARGON2_DWORDS_IN_BLOCK, thread, shared);
	}
}

KernelRunner::KernelRunner(uint32_t type, uint32_t version, uint32_t passes,
                           uint32_t lanes, uint32_t segmentBlocks,
                           size_t batchSize, size_t outLen, int32_t deviceIndex,
                           bool bySegment, bool precompute,
                           std::uint8_t *secret_, std::size_t secretLen_,
                           std::uint8_t *ad_, std::size_t adLen_)
    : type(type), version(version), passes(passes), lanes(lanes),
      segmentBlocks(segmentBlocks), batchSize(batchSize), outLen(outLen), bySegment(bySegment),
      precompute(precompute), stream(nullptr), memory(nullptr), seed(nullptr), seed_host(nullptr),
      out(nullptr), out_host(nullptr), refs(nullptr)
{

    int currentIndex = -1;
    CudaException::check(hipGetDevice(&currentIndex));
    if (currentIndex != deviceIndex) {
        CudaException::check(hipSetDevice(deviceIndex));
    }

    // FIXME: check overflow:
    size_t memorySize = batchSize * lanes * segmentBlocks
            * ARGON2_SYNC_POINTS * ARGON2_BLOCK_SIZE;

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << memorySize << " bytes for memory..."
                  << std::endl;
#endif

    CudaException::check(hipMalloc(&memory, memorySize));
    CudaException::check(hipMalloc(&seed, batchSize * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t)));
    CudaException::check(hipMalloc(&out, batchSize * outLen));
    CudaException::check(hipHostMalloc(&seed_host, batchSize * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t)));
    CudaException::check(hipHostMalloc(&out_host, batchSize * outLen));

    if(secret_ != NULL) {
        secretLen = secretLen_;
        CudaException::check(hipMalloc(&secret, secretLen));
        CudaException::check(hipMemcpy(secret, secret_, secretLen, hipMemcpyHostToDevice));
    }
    else {
        secretLen = 0;
        secret = NULL;
    }

    if(ad_ != NULL) {
        adLen = adLen_;
        CudaException::check(hipMalloc(&ad, adLen));
        CudaException::check(hipMemcpy(ad, ad_, adLen, hipMemcpyHostToDevice));
    }
    else {
        adLen = 0;
        ad = NULL;
    }

    CudaException::check(hipStreamCreate(&stream));

    if ((type == ARGON2_I || type == ARGON2_ID) && precompute) {
        uint32_t segments =
                type == ARGON2_ID
                ? lanes * (ARGON2_SYNC_POINTS / 2)
                : passes * lanes * ARGON2_SYNC_POINTS;

        size_t refsSize = segments * segmentBlocks * sizeof(struct ref);

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << refsSize << " bytes for refs..."
                  << std::endl;
#endif

        CudaException::check(hipMalloc(&refs, refsSize));

        precomputeRefs();
        synchronize();
    }
}

void KernelRunner::precomputeRefs()
{
    struct ref *refs = (struct ref *)this->refs;

    uint32_t segmentAddrBlocks = (segmentBlocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t segments =
            type == ARGON2_ID
            ? lanes * (ARGON2_SYNC_POINTS / 2)
            : passes * lanes * ARGON2_SYNC_POINTS;

    dim3 blocks = dim3(1, segments * segmentAddrBlocks);
    dim3 threads = dim3(THREADS_PER_LANE);

    size_t shmemSize = sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        argon2_precompute_kernel<ARGON2_I>
            <<<blocks, threads, shmemSize, stream>>>(
                refs, passes, lanes, segmentBlocks);
    } else {
        argon2_precompute_kernel<ARGON2_ID>
            <<<blocks, threads, shmemSize, stream>>>(
                refs, passes, lanes, segmentBlocks);
    }
}

KernelRunner::~KernelRunner()
{
    if (stream != nullptr) {
        hipStreamDestroy(stream);
    }
    if (memory != nullptr) {
        hipFree(memory);
    }
    if (refs != nullptr) {
        hipFree(refs);
    }
    if (seed != nullptr) {
        hipFree(seed);
    }
    if (out != nullptr) {
        hipFree(out);
    }
    if (seed_host != nullptr) {
        hipHostFree(seed_host);
    }
    if (out_host != nullptr) {
        hipHostFree(out_host);
    }
}

void *KernelRunner::getSeedBuffer(int index) {
    return &((uint8_t*)seed_host)[index * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t)];
}

void *KernelRunner::getOutBuffer(int index) {
    return &((uint8_t*)out_host)[index * outLen];
}

void KernelRunner::writeInputMemory(CoinAlgo algo)
{
    if(algo == None)
        CudaException::check(hipMemcpyAsync(seed, seed_host, batchSize * ARGON2_PREHASH_DIGEST_LENGTH * sizeof(uint32_t),
                                         hipMemcpyHostToDevice, stream));
    else
        CudaException::check(hipMemcpyAsync(seed, seed_host, 80,
                                             hipMemcpyHostToDevice, stream));
}

void KernelRunner::readOutputMemory()
{
    CudaException::check(hipMemcpyAsync(out_host, out, batchSize * outLen,
                                         hipMemcpyDeviceToHost, stream));
}

void KernelRunner::runKernelSegment(uint32_t lanesPerBlock,
                                    uint32_t jobsPerBlock,
                                    uint32_t pass, uint32_t slice)
{
    if (lanesPerBlock > lanes || lanes % lanesPerBlock != 0) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, lanes / lanesPerBlock, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanesPerBlock, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    uint32_t shared_size = blockSize * sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        } else {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        }
    }
}

void KernelRunner::runKernelOneshot(uint32_t lanesPerBlock,
                                    uint32_t jobsPerBlock)
{
    if (lanesPerBlock != lanes) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, 1, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanes, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    uint32_t shared_size = blockSize * sizeof(struct u64_shuffle_buf);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        } else {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        }
    }
}

void KernelRunner::runKernelPreseed(CoinAlgo algo) {
	struct block_g *memory_blocks = (struct block_g *)memory;
	argon2_kernel_preseed<<<batchSize, lanes * 8, lanes * 2 * BLAKE_SHARED_MEM>>>(algo, memory_blocks, (uint32_t *)seed, lanes, segmentBlocks, secret, secretLen, ad, adLen);
}

void KernelRunner::runKernelFinalize() {
	struct block_g *memory_blocks = (struct block_g *)memory;
	argon2_kernel_finalize<<<batchSize, 32, BLAKE_SHARED_MEM>>>(memory_blocks, (uint32_t *)out, outLen / 4, lanes, segmentBlocks);
}

void KernelRunner::run(CoinAlgo algo, uint32_t lanesPerBlock, uint32_t jobsPerBlock)
{
    timer = get_time();
    runKernelPreseed(algo);
    if (bySegment) {
        for (uint32_t pass = 0; pass < passes; pass++) {
            for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; slice++) {
                runKernelSegment(lanesPerBlock, jobsPerBlock, pass, slice);
            }
        }
    } else {
        runKernelOneshot(lanesPerBlock, jobsPerBlock);
    }
	runKernelFinalize();
}

uint64_t KernelRunner::finish()
{
	synchronize();
	return get_time() - timer;
}

void KernelRunner::synchronize() {
	while(hipStreamQuery(stream) != hipSuccess) {
		std::this_thread::sleep_for(std::chrono::milliseconds(1));
	};

	CudaException::check(hipGetLastError());
}

uint64_t KernelRunner::get_time() {
#ifdef _WIN64
	FILETIME ft;
	uint64_t tmpres = 0;
	static int tzflag;

	GetSystemTimeAsFileTime(&ft);

	tmpres |= ft.dwHighDateTime;
	tmpres <<= 32;
	tmpres |= ft.dwLowDateTime;

	return tmpres;
#else
    timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return t.tv_sec * 1000000000 + t.tv_nsec;
#endif
}

} // cuda
} // argon2
